#include <Operations.cuh>
#include <Slab.cuh>

#define PAGE_SIZE 4096

SlabUnified::SlabUnified(int size) : SlabUnified(size, 0, nullptr) {}

SlabUnified::SlabUnified(int size, int gpu)
        : SlabUnified(size, gpu, nullptr) {}

SlabUnified::SlabUnified(int size, hipStream_t *stream)
        : SlabUnified(size, 0, stream) {}

SlabUnified::SlabUnified(int size, int gpu, hipStream_t *stream) {
    gpuErrchk(hipSetDevice(gpu));
    slabGAlloc = new groupallocator::GroupAllocator(0, PAGE_SIZE);
    allocGAlloc = new groupallocator::GroupAllocator(1, PAGE_SIZE);
    bufferGAlloc = new groupallocator::GroupAllocator(2, 4096);
    this->slab = setUpGroup(*slabGAlloc, size, 1, gpu, (stream == nullptr ? hipStreamDefault : *stream));

#ifdef USE_HOST
    gpuErrchk(hipMalloc(&batchKeys, BLOCKS * THREADS_PER_BLOCK * sizeof(unsigned)));
    gpuErrchk(hipMalloc(&batchValues, BLOCKS * THREADS_PER_BLOCK * sizeof(unsigned)));
    gpuErrchk(hipMalloc(&batchRequests, BLOCKS * THREADS_PER_BLOCK * sizeof(int)));
    gpuErrchk(hipHostMalloc(&batchKeys_h, BLOCKS * THREADS_PER_BLOCK * sizeof(unsigned)));
    gpuErrchk(hipHostMalloc(&batchValues_h, BLOCKS * THREADS_PER_BLOCK * sizeof(unsigned)));
    gpuErrchk(hipHostMalloc(&batchRequests_h, BLOCKS * THREADS_PER_BLOCK * sizeof(int)));

#else
    bufferGAlloc->allocate(&batchKeys,
                           BLOCKS * THREADS_PER_BLOCK * sizeof(unsigned), false);
    bufferGAlloc->allocate(&batchValues,
                           BLOCKS * THREADS_PER_BLOCK * sizeof(unsigned), false);
    bufferGAlloc->allocate(&batchRequests,
                           BLOCKS * THREADS_PER_BLOCK * sizeof(int), false);
#endif

    this->ctx = setupWarpAllocCtxGroup(*allocGAlloc, THREADS_PER_BLOCK, BLOCKS,
                                       gpu, (stream == nullptr ? hipStreamDefault : *stream));
    _stream = stream;
    _gpu = gpu;
    mapSize = size;
}

SlabUnified::~SlabUnified() {
    delete slabGAlloc;
    delete allocGAlloc;
    delete bufferGAlloc;

#ifdef USE_HOST
    gpuErrchk(hipFree(batchKeys));
    gpuErrchk(hipFree(batchValues));
    gpuErrchk(hipFree(batchRequests));
    gpuErrchk(hipHostFree(batchKeys_h));
    gpuErrchk(hipHostFree(batchValues_h));
    gpuErrchk(hipHostFree(batchRequests_h));

#endif

}

void SlabUnified::batch(unsigned *keys, unsigned *values, unsigned *requests) {

    gpuErrchk(hipSetDevice(_gpu));

#ifdef USE_HOST
    for (int i = 0; i < THREADS_PER_BLOCK * BLOCKS; i++) {
        batchKeys_h[i] = keys[i];
        batchValues_h[i] = values[i];
        batchRequests_h[i] = requests[i];
    }

    auto stream = (_stream == nullptr ? hipStreamDefault : *_stream);

    gpuErrchk(hipMemcpyAsync(batchKeys, batchKeys_h, BLOCKS * THREADS_PER_BLOCK * sizeof(unsigned), hipMemcpyHostToDevice, stream));
    gpuErrchk(hipMemcpyAsync(batchValues, batchValues_h, BLOCKS * THREADS_PER_BLOCK * sizeof(unsigned), hipMemcpyHostToDevice, stream));
    gpuErrchk(hipMemcpyAsync(batchRequests, batchRequests_h, BLOCKS * THREADS_PER_BLOCK * sizeof(int), hipMemcpyHostToDevice, stream));

    gpuErrchk(hipStreamSynchronize(stream));

#else
    for (int i = 0; i < THREADS_PER_BLOCK * BLOCKS; i++) {
        batchKeys[i] = keys[i];
        batchValues[i] = values[i];
        batchRequests[i] = requests[i];
    }

    auto stream = (_stream == nullptr ? hipStreamDefault : *_stream);

    bufferGAlloc->moveToDevice(_gpu, stream);
    gpuErrchk(hipStreamSynchronize(stream));
#endif

    requestHandler<<<BLOCKS, THREADS_PER_BLOCK, 0, stream>>>(
            slab->slabs, slab->num_of_buckets, batchKeys, batchValues, batchRequests,
            ctx);
    gpuErrchk(hipStreamSynchronize(stream));

#ifdef USE_HOST

    gpuErrchk(hipMemcpyAsync(batchKeys_h, batchKeys, BLOCKS * THREADS_PER_BLOCK * sizeof(unsigned), hipMemcpyDeviceToHost, stream));
    gpuErrchk(hipMemcpyAsync(batchValues_h, batchValues, BLOCKS * THREADS_PER_BLOCK * sizeof(unsigned), hipMemcpyDeviceToHost, stream));
    gpuErrchk(hipMemcpyAsync(batchRequests_h, batchRequests, BLOCKS * THREADS_PER_BLOCK * sizeof(int), hipMemcpyDeviceToHost, stream));

    gpuErrchk(hipStreamSynchronize(stream));

    for (int i = 0; i < THREADS_PER_BLOCK * BLOCKS; i++) {
        keys[i] = batchKeys_h[i];
        values[i] = batchValues_h[i];
        requests[i] = batchRequests_h[i];
    }

#else
    bufferGAlloc->moveToDevice(hipCpuDeviceId, stream);
    gpuErrchk(hipStreamSynchronize(stream));

    for (int i = 0; i < THREADS_PER_BLOCK * BLOCKS; i++) {
        keys[i] = batchKeys[i];
        values[i] = batchValues[i];
        requests[i] = batchRequests[i];
    }
#endif
}

std::tuple<float, float, float> SlabUnified::batch_bench(unsigned *keys, unsigned *values, unsigned *requests) {
    gpuErrchk(hipSetDevice(_gpu));

#ifdef USE_HOST
    for (int i = 0; i < THREADS_PER_BLOCK * BLOCKS; i++) {
        batchKeys_h[i] = keys[i];
        batchValues_h[i] = values[i];
        batchRequests_h[i] = requests[i];
    }

#else

    for (int i = 0; i < THREADS_PER_BLOCK * BLOCKS; i++) {
        batchKeys[i] = keys[i];
        batchValues[i] = values[i];
        batchRequests[i] = requests[i];
    }

#endif
    auto stream = (_stream == nullptr ? hipStreamDefault : *_stream);

    hipEvent_t start_m, start_k, end_k, end_m;

    gpuErrchk(hipEventCreate(&start_m));
    gpuErrchk(hipEventCreate(&start_k));

    gpuErrchk(hipEventCreate(&end_k));
    gpuErrchk(hipEventCreate(&end_m));

    hipEventRecord(start_m, stream);
#ifdef USE_HOST
    gpuErrchk(hipMemcpyAsync(batchKeys, batchKeys_h, BLOCKS * THREADS_PER_BLOCK * sizeof(unsigned), hipMemcpyHostToDevice, stream));
    gpuErrchk(hipMemcpyAsync(batchValues, batchValues_h, BLOCKS * THREADS_PER_BLOCK * sizeof(unsigned), hipMemcpyHostToDevice, stream));
    gpuErrchk(hipMemcpyAsync(batchRequests, batchRequests_h, BLOCKS * THREADS_PER_BLOCK * sizeof(int), hipMemcpyHostToDevice, stream));
#else
    bufferGAlloc->moveToDevice(_gpu, stream);
#endif
    hipEventRecord(start_k, stream);
    requestHandler<<<BLOCKS, THREADS_PER_BLOCK, 0, stream>>>(
            slab->slabs, slab->num_of_buckets, batchKeys, batchValues, batchRequests,
            ctx);
    hipEventRecord(end_k, stream);
#ifdef USE_HOST
    gpuErrchk(hipMemcpyAsync(batchKeys_h, batchKeys, BLOCKS * THREADS_PER_BLOCK * sizeof(unsigned), hipMemcpyDeviceToHost, stream));
    gpuErrchk(hipMemcpyAsync(batchValues_h, batchValues, BLOCKS * THREADS_PER_BLOCK * sizeof(unsigned), hipMemcpyDeviceToHost, stream));
    gpuErrchk(hipMemcpyAsync(batchRequests_h, batchRequests, BLOCKS * THREADS_PER_BLOCK * sizeof(int), hipMemcpyDeviceToHost, stream));
#else
    bufferGAlloc->moveToDevice(hipCpuDeviceId, stream);
#endif
    hipEventRecord(end_m, stream);

    gpuErrchk(hipEventSynchronize(end_m));

    float ms1, ms2, ms3;
    gpuErrchk(hipEventElapsedTime(&ms1, start_k, end_k));
    gpuErrchk(hipEventElapsedTime(&ms2, start_m, start_k));
    gpuErrchk(hipEventElapsedTime(&ms3, end_k, end_m));


    gpuErrchk(hipEventDestroy(start_m));
    gpuErrchk(hipEventDestroy(start_k));

    gpuErrchk(hipEventDestroy(end_k));
    gpuErrchk(hipEventDestroy(end_m));

#ifdef USE_HOST
    for (int i = 0; i < THREADS_PER_BLOCK * BLOCKS; i++) {
        keys[i] = batchKeys_h[i];
        values[i] = batchValues_h[i];
        requests[i] = batchRequests_h[i];
    }
#else
    for (int i = 0; i < THREADS_PER_BLOCK * BLOCKS; i++) {
        keys[i] = batchKeys[i];
        values[i] = batchValues[i];
        requests[i] = batchRequests[i];
    }
#endif
    return {ms1, ms2, ms3};
}

std::tuple<float, float, float> SlabUnified::batch_get(unsigned *keys, unsigned *values, unsigned *requests) {
    gpuErrchk(hipSetDevice(_gpu));

#ifdef USE_HOST
    for (int i = 0; i < THREADS_PER_BLOCK * BLOCKS; i++) {
        batchKeys_h[i] = keys[i];
        batchValues_h[i] = values[i];
        batchRequests_h[i] = requests[i];
    }

#else

    for (int i = 0; i < THREADS_PER_BLOCK * BLOCKS; i++) {
        batchKeys[i] = keys[i];
        batchValues[i] = values[i];
        batchRequests[i] = requests[i];
    }

#endif
    auto stream = (_stream == nullptr ? hipStreamDefault : *_stream);

    hipEvent_t start_m, start_k, end_k, end_m;

    gpuErrchk(hipEventCreate(&start_m));
    gpuErrchk(hipEventCreate(&start_k));

    gpuErrchk(hipEventCreate(&end_k));
    gpuErrchk(hipEventCreate(&end_m));

    hipEventRecord(start_m, stream);
#ifdef USE_HOST
    gpuErrchk(hipMemcpyAsync(batchKeys, batchKeys_h, BLOCKS * THREADS_PER_BLOCK * sizeof(unsigned), hipMemcpyHostToDevice, stream));
    gpuErrchk(hipMemcpyAsync(batchValues, batchValues_h, BLOCKS * THREADS_PER_BLOCK * sizeof(unsigned), hipMemcpyHostToDevice, stream));
    gpuErrchk(hipMemcpyAsync(batchRequests, batchRequests_h, BLOCKS * THREADS_PER_BLOCK * sizeof(int), hipMemcpyHostToDevice, stream));
#else
    bufferGAlloc->moveToDevice(_gpu, stream);
#endif
    hipEventRecord(start_k, stream);
    getHandler<<<BLOCKS, THREADS_PER_BLOCK, 0, stream>>>(
            slab->slabs, slab->num_of_buckets, batchKeys, batchValues, batchRequests);
    hipEventRecord(end_k, stream);
#ifdef USE_HOST
    gpuErrchk(hipMemcpyAsync(batchKeys_h, batchKeys, BLOCKS * THREADS_PER_BLOCK * sizeof(unsigned), hipMemcpyDeviceToHost, stream));
    gpuErrchk(hipMemcpyAsync(batchValues_h, batchValues, BLOCKS * THREADS_PER_BLOCK * sizeof(unsigned), hipMemcpyDeviceToHost, stream));
    gpuErrchk(hipMemcpyAsync(batchRequests_h, batchRequests, BLOCKS * THREADS_PER_BLOCK * sizeof(int), hipMemcpyDeviceToHost, stream));
#else
    bufferGAlloc->moveToDevice(hipCpuDeviceId, stream);
#endif
    hipEventRecord(end_m, stream);

    gpuErrchk(hipEventSynchronize(end_m));

    float ms1, ms2, ms3;
    gpuErrchk(hipEventElapsedTime(&ms1, start_k, end_k));
    gpuErrchk(hipEventElapsedTime(&ms2, start_m, start_k));
    gpuErrchk(hipEventElapsedTime(&ms3, end_k, end_m));


    gpuErrchk(hipEventDestroy(start_m));
    gpuErrchk(hipEventDestroy(start_k));

    gpuErrchk(hipEventDestroy(end_k));
    gpuErrchk(hipEventDestroy(end_m));

#ifdef USE_HOST
    for (int i = 0; i < THREADS_PER_BLOCK * BLOCKS; i++) {
        keys[i] = batchKeys_h[i];
        values[i] = batchValues_h[i];
        requests[i] = batchRequests_h[i];
    }
#else
    for (int i = 0; i < THREADS_PER_BLOCK * BLOCKS; i++) {
        keys[i] = batchKeys[i];
        values[i] = batchValues[i];
        requests[i] = batchRequests[i];
    }
#endif
    return {ms1, ms2, ms3};
}

std::tuple<float, float, float> SlabUnified::batch_insert(unsigned *keys, unsigned *values, unsigned *requests) {
    gpuErrchk(hipSetDevice(_gpu));

#ifdef USE_HOST
    for (int i = 0; i < THREADS_PER_BLOCK * BLOCKS; i++) {
        batchKeys_h[i] = keys[i];
        batchValues_h[i] = values[i];
        batchRequests_h[i] = requests[i];
    }

#else

    for (int i = 0; i < THREADS_PER_BLOCK * BLOCKS; i++) {
        batchKeys[i] = keys[i];
        batchValues[i] = values[i];
        batchRequests[i] = requests[i];
    }

#endif
    auto stream = (_stream == nullptr ? hipStreamDefault : *_stream);

    hipEvent_t start_m, start_k, end_k, end_m;

    gpuErrchk(hipEventCreate(&start_m));
    gpuErrchk(hipEventCreate(&start_k));

    gpuErrchk(hipEventCreate(&end_k));
    gpuErrchk(hipEventCreate(&end_m));

    hipEventRecord(start_m, stream);
#ifdef USE_HOST
    gpuErrchk(hipMemcpyAsync(batchKeys, batchKeys_h, BLOCKS * THREADS_PER_BLOCK * sizeof(unsigned), hipMemcpyHostToDevice, stream));
    gpuErrchk(hipMemcpyAsync(batchValues, batchValues_h, BLOCKS * THREADS_PER_BLOCK * sizeof(unsigned), hipMemcpyHostToDevice, stream));
    gpuErrchk(hipMemcpyAsync(batchRequests, batchRequests_h, BLOCKS * THREADS_PER_BLOCK * sizeof(int), hipMemcpyHostToDevice, stream));
#else
    bufferGAlloc->moveToDevice(_gpu, stream);
#endif
    hipEventRecord(start_k, stream);
    insertHandler<<<BLOCKS, THREADS_PER_BLOCK, 0, stream>>>(
            slab->slabs, slab->num_of_buckets, batchKeys, batchValues, batchRequests,
            ctx);
    hipEventRecord(end_k, stream);
#ifdef USE_HOST
    gpuErrchk(hipMemcpyAsync(batchKeys_h, batchKeys, BLOCKS * THREADS_PER_BLOCK * sizeof(unsigned), hipMemcpyDeviceToHost, stream));
    gpuErrchk(hipMemcpyAsync(batchValues_h, batchValues, BLOCKS * THREADS_PER_BLOCK * sizeof(unsigned), hipMemcpyDeviceToHost, stream));
    gpuErrchk(hipMemcpyAsync(batchRequests_h, batchRequests, BLOCKS * THREADS_PER_BLOCK * sizeof(int), hipMemcpyDeviceToHost, stream));
#else
    bufferGAlloc->moveToDevice(hipCpuDeviceId, stream);
#endif
    hipEventRecord(end_m, stream);

    gpuErrchk(hipEventSynchronize(end_m));

    float ms1, ms2, ms3;
    gpuErrchk(hipEventElapsedTime(&ms1, start_k, end_k));
    gpuErrchk(hipEventElapsedTime(&ms2, start_m, start_k));
    gpuErrchk(hipEventElapsedTime(&ms3, end_k, end_m));


    gpuErrchk(hipEventDestroy(start_m));
    gpuErrchk(hipEventDestroy(start_k));

    gpuErrchk(hipEventDestroy(end_k));
    gpuErrchk(hipEventDestroy(end_m));

#ifdef USE_HOST
    for (int i = 0; i < THREADS_PER_BLOCK * BLOCKS; i++) {
        keys[i] = batchKeys_h[i];
        values[i] = batchValues_h[i];
        requests[i] = batchRequests_h[i];
    }
#else
    for (int i = 0; i < THREADS_PER_BLOCK * BLOCKS; i++) {
        keys[i] = batchKeys[i];
        values[i] = batchValues[i];
        requests[i] = batchRequests[i];
    }
#endif
    return {ms1, ms2, ms3};
}

std::tuple<float, float, float> SlabUnified::batch_delete(unsigned *keys, unsigned *values, unsigned *requests) {
    gpuErrchk(hipSetDevice(_gpu));

#ifdef USE_HOST
    for (int i = 0; i < THREADS_PER_BLOCK * BLOCKS; i++) {
        batchKeys_h[i] = keys[i];
        batchValues_h[i] = values[i];
        batchRequests_h[i] = requests[i];
    }

#else

    for (int i = 0; i < THREADS_PER_BLOCK * BLOCKS; i++) {
        batchKeys[i] = keys[i];
        batchValues[i] = values[i];
        batchRequests[i] = requests[i];
    }

#endif
    auto stream = (_stream == nullptr ? hipStreamDefault : *_stream);

    hipEvent_t start_m, start_k, end_k, end_m;

    gpuErrchk(hipEventCreate(&start_m));
    gpuErrchk(hipEventCreate(&start_k));

    gpuErrchk(hipEventCreate(&end_k));
    gpuErrchk(hipEventCreate(&end_m));

    hipEventRecord(start_m, stream);
#ifdef USE_HOST
    gpuErrchk(hipMemcpyAsync(batchKeys, batchKeys_h, BLOCKS * THREADS_PER_BLOCK * sizeof(unsigned), hipMemcpyHostToDevice, stream));
    gpuErrchk(hipMemcpyAsync(batchValues, batchValues_h, BLOCKS * THREADS_PER_BLOCK * sizeof(unsigned), hipMemcpyHostToDevice, stream));
    gpuErrchk(hipMemcpyAsync(batchRequests, batchRequests_h, BLOCKS * THREADS_PER_BLOCK * sizeof(int), hipMemcpyHostToDevice, stream));
#else
    bufferGAlloc->moveToDevice(_gpu, stream);
#endif
    hipEventRecord(start_k, stream);
    deleteHandler<<<BLOCKS, THREADS_PER_BLOCK, 0, stream>>>(
            slab->slabs, slab->num_of_buckets, batchKeys, batchValues, batchRequests);
    hipEventRecord(end_k, stream);
#ifdef USE_HOST
    gpuErrchk(hipMemcpyAsync(batchKeys_h, batchKeys, BLOCKS * THREADS_PER_BLOCK * sizeof(unsigned), hipMemcpyDeviceToHost, stream));
    gpuErrchk(hipMemcpyAsync(batchValues_h, batchValues, BLOCKS * THREADS_PER_BLOCK * sizeof(unsigned), hipMemcpyDeviceToHost, stream));
    gpuErrchk(hipMemcpyAsync(batchRequests_h, batchRequests, BLOCKS * THREADS_PER_BLOCK * sizeof(int), hipMemcpyDeviceToHost, stream));
#else
    bufferGAlloc->moveToDevice(hipCpuDeviceId, stream);
#endif
    hipEventRecord(end_m, stream);

    gpuErrchk(hipEventSynchronize(end_m));

    float ms1, ms2, ms3;
    gpuErrchk(hipEventElapsedTime(&ms1, start_k, end_k));
    gpuErrchk(hipEventElapsedTime(&ms2, start_m, start_k));
    gpuErrchk(hipEventElapsedTime(&ms3, end_k, end_m));


    gpuErrchk(hipEventDestroy(start_m));
    gpuErrchk(hipEventDestroy(start_k));

    gpuErrchk(hipEventDestroy(end_k));
    gpuErrchk(hipEventDestroy(end_m));

#ifdef USE_HOST
    for (int i = 0; i < THREADS_PER_BLOCK * BLOCKS; i++) {
        keys[i] = batchKeys_h[i];
        values[i] = batchValues_h[i];
        requests[i] = batchRequests_h[i];
    }
#else
    for (int i = 0; i < THREADS_PER_BLOCK * BLOCKS; i++) {
        keys[i] = batchKeys[i];
        values[i] = batchValues[i];
        requests[i] = batchRequests[i];
    }
#endif
    return {ms1, ms2, ms3};
}
