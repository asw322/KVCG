#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2015 Kai Zhang (kay21s@gmail.com)
 *
 * Permission is hereby granted, free of charge, to any person obtaining a copy
 * of this software and associated documentation files (the "Software"), to deal
 * in the Software without restriction, including without limitation the rights
 * to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
 * copies of the Software, and to permit persons to whom the Software is
 * furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in
 * all copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
 * AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
 * OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN
 * THE SOFTWARE.
 */

#include <stdint.h>
#include <stdio.h>
#include <assert.h>
#include "gpu_hash.h"

#define FULL_MASK 0xffffffff

__global__ void hash_search(
		selem_t			*in,
		loc_t			*out,
		bucket_t		*hash_table,
		int				total_elem_num,
		int				thread_num)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x; 
	int id = 0;
	// (1 << ELEM_NUM_P) threads to cooperate for one element
	int step = thread_num >> ELEM_NUM_P;
	int ballot;
	
	int simd_lane = idx & ((1 << ELEM_NUM_P) - 1);
	int elem_id = idx >> ELEM_NUM_P;

	int bit_move;
	bit_move = idx & (((1 << (5 - ELEM_NUM_P)) - 1) << ELEM_NUM_P);

	for (id = elem_id; id < total_elem_num; id += step) {
		selem_t *elem = &(in[id]);

		// TODO: force out buffer to be memset zero so that
		// this simultaneous memory write can be avoided
		// out[id << 1] = 0;
		// out[id << 1 + 1] = 0;

		bucket_t *b = &(hash_table[elem->hash & HASH_MASK]);
		if (b->sig[simd_lane] == elem->sig) {
			out[id << 1] = b->loc[simd_lane];
		}
		ballot = __ballot_sync(FULL_MASK, b->sig[simd_lane] == elem->sig);
		ballot = (ballot >> bit_move) & ((1 << ELEM_NUM) - 1);
		//if (ballot != 0) {
		//	continue;
		//}

		//b = &(hash_table[(elem->hash ^ elem->sig) & HASH_MASK]);
		int hash = (((elem->hash ^ elem->sig) & BLOCK_HASH_MASK) 
				| (elem->hash & ~BLOCK_HASH_MASK)) & HASH_MASK; 
		b = &(hash_table[hash]);
		if (b->sig[simd_lane] == elem->sig) {
			out[(id << 1) + 1] = b->loc[simd_lane];
		}
	}

	return;
}

__global__ void hash_insert_2choice(
		bucket_t		*hash_table,
		ielem_t			**blk_input,
		int				*blk_elem_num)
{
	ielem_t *in = blk_input[blockIdx.x];
	int total_elem_num = blk_elem_num[blockIdx.x];
	// 16 threads to cooperate for one element
	int step = blockDim.x >> ELEM_NUM_P;
	int idx = threadIdx.x;

	int id = 0, hash;
	bucket_t *b;
	int chosen_simd;
	int ballot, ml_mask;

	int simd_lane = idx & ((1 << ELEM_NUM_P) - 1);
	int elem_id = idx >> ELEM_NUM_P;
	int bit_move = idx & (((1 << (5 - ELEM_NUM_P)) - 1) << ELEM_NUM_P);

	for (id = elem_id; id < total_elem_num; id += step) {
		ielem_t *elem = &(in[id]);

		sign_t sig = elem->sig;
		if (elem->sig == 0 && elem->loc == 0) {
			printf("error, all is zero\n");
			continue;
		}

		b = &(hash_table[elem->hash & HASH_MASK]);

		/*=====================================================================
		 * The double __syncthreads() seems useless in else, this is to match the two in
		 * if (chosen_simd == simd_lane). As is stated in the paper <Demystifying GPU 
		 * Microarchitecture through Microbenchmarking>, the __syncthreads() will not go
		 * wrong if not all threads in one wrap reach it, however, the wraps in the same
		 * block need to reach a __syncthreads(), even if they are not on the same line */
		/* Check for same signatures in two bucket */
		ballot = __ballot_sync(FULL_MASK, b->sig[simd_lane] == elem->sig);
		/* first half warp(0~15 threads), bit_move = 0
		 * for bottom half warp(16~31 threads), bit_move = 16 */
		ballot = (ballot >> bit_move) & ((1 << ELEM_NUM) - 1);
		if (0 != ballot) {
			chosen_simd = (__ffs(ballot) - 1) & ((1 << ELEM_NUM_P) - 1);
			if (simd_lane == chosen_simd) {
				b->loc[simd_lane] = elem->loc;
			}
			continue;
		}

		/*=====================================================================*/
		/* Next we try to insert, the while look breaks if ballot == 0, and the 
		 * __syncthreads() in the two loops match if the code path divergent between
		 * the warps in a block. Or some will terminate, or process the next element. 
		 * FIXME: if some wrap go to process next element, some stays here, will this
		 * lead to mismatch in __syncthreads()? If it does, we should launch one thread
		 * for each element. God knows what nVidia GPU will behave. FIXME;
		 * Here we write b->loc, and the above code also write b->loc. This will not
		 * lead to conflicts, because here all the signatures are 0, while the aboves
		 * are all non-zero */

		/* Major Location : use last 4 bits of signature */
		ml_mask = (1 << (elem->sig & ((1 << ELEM_NUM_P) - 1))) - 1;
		/* find the empty slot for insertion */
		while (1) {
			ballot = __ballot_sync(FULL_MASK, b->sig[simd_lane] == 0);
			ballot = (ballot >> bit_move) & ((1 << ELEM_NUM) - 1);
			/* 1010|0011 => 0000 0011 1010 0000, 16 bits to 32 bits*/
			ballot = ((ballot & ml_mask) << 16) | ((ballot & ~(ml_mask)));
			if (ballot != 0) {
				chosen_simd = (__ffs(ballot) - 1) & ((1 << ELEM_NUM_P) - 1);
				if (simd_lane == chosen_simd) {
					b->sig[simd_lane] = sig;
				}
			}

			__syncthreads();

			if (ballot != 0) {
				if (b->sig[chosen_simd] == sig) {
					if (simd_lane == chosen_simd) {
						b->loc[simd_lane] = elem->loc;
					}
					goto finish;
				}
			} else {
				break;
			}
		}


		/* ==== try next bucket ==== */


		hash = (((elem->hash ^ sig) & BLOCK_HASH_MASK) 
				| (elem->hash & ~BLOCK_HASH_MASK)) & HASH_MASK; 
		b = &(hash_table[hash]);
		/*=====================================================================*/
		/* Check for same signatures in two bucket */
		ballot = __ballot_sync(FULL_MASK, b->sig[simd_lane] == elem->sig);
		/* first half warp(0~15 threads), bit_move = 0
		 * for bottom half warp(16~31 threads), bit_move = 16 */
		ballot = (ballot >> bit_move) & ((1 << ELEM_NUM) - 1);
		if (0 != ballot) {
			chosen_simd = (__ffs(ballot) - 1) & ((1 << ELEM_NUM_P) - 1);
			if (simd_lane == chosen_simd) {
				b->loc[simd_lane] = elem->loc;
			}
			continue;
		}

		while (1) {
			ballot = __ballot_sync(FULL_MASK, b->sig[simd_lane] == 0);
			ballot = (ballot >> bit_move) & ((1 << ELEM_NUM) - 1);
			ballot = ((ballot & ml_mask) << 16) | ((ballot & ~(ml_mask)));
			if (ballot != 0) {
				chosen_simd = (__ffs(ballot) - 1) & ((1 << ELEM_NUM_P) - 1);
				if (simd_lane == chosen_simd) {
					b->sig[simd_lane] = sig;
				}
			} else {
				/* No available slot.
				 * Get a Major location between 0 and 15 for insertion */
				chosen_simd = elem->sig & ((1 << ELEM_NUM_P) - 1);
				if (simd_lane == chosen_simd) {
					b->sig[simd_lane] = sig;
				}
				/* we only try insert once if there are no empty slots,
				 * because conflicted items on the same chosen_simd will
				 * keep conflicting. 
				 */
				break;
			}

			__syncthreads();

			/* chosen_simd controls one thread in a half warp
			 * enters this */
			if (b->sig[chosen_simd] == sig) {
				if (simd_lane == chosen_simd) {
					b->loc[simd_lane] = elem->loc;
				}
				goto finish;
			}
		}

finish:
		;
		//now we get to the next element
	}

	return;
}

#define BARRIER() __syncwarp()

__global__ void hash_insert_cuckoo(
		bucket_t		*hash_table,
		ielem_t			**blk_input,
		int				*blk_elem_num)
{
	ielem_t *in = blk_input[blockIdx.x];
	int total_elem_num = blk_elem_num[blockIdx.x];
	// 16 threads to cooperate for one element
	int step = blockDim.x >> ELEM_NUM_P;
	int idx = threadIdx.x;

	hash_t hash, second_hash;
	loc_t loc, new_loc;
	sign_t sig, new_sig;

	int id;
	int cuckoo_num;
	bucket_t *b;
	int chosen_simd;
	int ballot, ml_mask;

	int simd_lane = idx & ((1 << ELEM_NUM_P) - 1);
	int elem_id = idx >> ELEM_NUM_P;
	int bit_move = idx & (((1 << (5 - ELEM_NUM_P)) - 1) << ELEM_NUM_P);

	for (id = elem_id; id < total_elem_num; id += step) {
		ielem_t *elem = &(in[id]);

		if (elem->sig == 0 && elem->loc == 0) {
			printf("error, all is zero\n");
			continue;
		}

		sig = elem->sig;
		hash = elem->hash;
		loc = elem->loc;

		b = &(hash_table[hash & HASH_MASK]);

		/*=====================================================================
		 * The double __syncthreads() seems useless in else, this is to match the two in
		 * if (chosen_simd == simd_lane). As is stated in the paper <Demystifying GPU 
		 * Microarchitecture through Microbenchmarking>, the __syncthreads() will not go
		 * wrong if not all threads in one wrap reach it. However, the wraps in the same
		 * block need to reach a __syncthreads(), even if they are not on the same line */
		/* Check for same signatures in two bucket */
		ballot = __ballot_sync(FULL_MASK, b->sig[simd_lane] == sig);
		/* first half warp(0~15 threads), bit_move = 0
		 * for second half warp(16~31 threads), bit_move = 16 */
		ballot = (ballot >> bit_move) & ((1 << ELEM_NUM) - 1);
		if (ballot != 0) {
			chosen_simd = (__ffs(ballot) - 1) & ((1 << ELEM_NUM_P) - 1);
			if (simd_lane == chosen_simd) {
				b->loc[simd_lane] = loc;
			}
			continue;
		}

		/*=====================================================================*/
		/* Next we try to insert, the while loop breaks if ballot == 0, and the 
		 * __syncthreads() in the two loops match if the code path divergent between
		 * the warps in a block. Or some will terminate, or process the next element. 
		 * FIXME: if some wrap go to process next element, some stays here, will this
		 * lead to mismatch in __syncthreads()? If it does, we should launch one thread
		 * for each element. God knows what nVidia GPU will behave. FIXME;
		 * Here we write b->loc, and the above code also write b->loc. This will not
		 * lead to conflicts, because here all the signatures are 0, while the aboves
		 * are all non-zero */

		/* Major Location : use last 4 bits of signature */
		ml_mask = (1 << (sig & ((1 << ELEM_NUM_P) - 1))) - 1;
		/* find the empty slot for insertion */
		while (1) {
			ballot = __ballot_sync(FULL_MASK, b->sig[simd_lane] == 0);
			ballot = (ballot >> bit_move) & ((1 << ELEM_NUM) - 1);
			/* 1010|0011 => 0000 0011 1010 0000, 16 bits to 32 bits*/
			ballot = ((ballot & ml_mask) << 16) | ((ballot & ~(ml_mask)));
			if (ballot != 0) {
				chosen_simd = (__ffs(ballot) - 1) & ((1 << ELEM_NUM_P) - 1);
				if (simd_lane == chosen_simd) {
					b->sig[simd_lane] = sig;
				}

            }

            BARRIER();

			if (ballot != 0) {
				if (b->sig[chosen_simd] == sig) {
					if (simd_lane == chosen_simd) {
						b->loc[simd_lane] = loc;
                    }
					goto finish;
				}
			} else {
				break;
			}
		}


		/* ==== try next bucket ==== */
		cuckoo_num = 0;

cuckoo_evict:
		second_hash = (((hash ^ sig) & BLOCK_HASH_MASK) 
				| (hash & ~BLOCK_HASH_MASK)) & HASH_MASK; 
		b = &(hash_table[second_hash]);
		/*=====================================================================*/
		/* Check for same signatures in two bucket */
		ballot = __ballot_sync(FULL_MASK, b->sig[simd_lane] == sig);
		/* first half warp(0~15 threads), bit_move = 0
		 * for second half warp(16~31 threads), bit_move = 16 */
		ballot = (ballot >> bit_move) & ((1 << ELEM_NUM) - 1);
		if (0 != ballot) {
			chosen_simd = (__ffs(ballot) - 1) & ((1 << ELEM_NUM_P) - 1);
			if (simd_lane == chosen_simd) {
				b->loc[simd_lane] = loc;

            }
			continue;
		}

		while (1) {
			ballot = __ballot_sync(FULL_MASK, b->sig[simd_lane] == 0);
			ballot = (ballot >> bit_move) & ((1 << ELEM_NUM) - 1);
			ballot = ((ballot & ml_mask) << 16) | ((ballot & ~(ml_mask)));
			if (ballot != 0) {
				chosen_simd = (__ffs(ballot) - 1) & ((1 << ELEM_NUM_P) - 1);
			} else {
				/* No available slot.
				 * Get a Major location between 0 and 15 for insertion */
				chosen_simd = elem->sig & ((1 << ELEM_NUM_P) - 1);
				if (cuckoo_num < MAX_CUCKOO_NUM) {
					/* record the signature to be evicted */
					new_sig = b->sig[chosen_simd];
					new_loc = b->loc[chosen_simd];
				}
			}
			
			/* synchronize before the signature is written by others */
            BARRIER();

			if (ballot != 0) {
				if (simd_lane == chosen_simd) {
					b->sig[simd_lane] = sig;

                }
			} else {
				/* two situations to handle: 1) cuckoo_num < MAX_CUCKOO_NUM,
				 * replace one element, and reinsert it into its alternative bucket.
				 * 2) cuckoo_num >= MAX_CUCKOO_NUM.
				 * The cuckoo evict exceed the maximum insert time, replace the element.
				 * In each case, we write the signature first.*/
				if (simd_lane == chosen_simd) {
					b->sig[simd_lane] = sig;

                }
			}

            BARRIER();

			if (ballot != 0) {
				/* write the empty slot or try again when conflict */
				if (b->sig[chosen_simd] == sig) {
					if (simd_lane == chosen_simd) {
						b->loc[simd_lane] = loc;

                    }
					goto finish;
				}
			} else {
				if (cuckoo_num < MAX_CUCKOO_NUM) {
					cuckoo_num ++;
					if (b->sig[chosen_simd] == sig) {
						if (simd_lane == chosen_simd) {
							b->loc[simd_lane] = loc;

                        }
						sig = new_sig;
						loc = new_loc;
						goto cuckoo_evict;
					} else {
						/* if there is conflict when writing the signature,
						 * it has been replaced by another one. Reinserting
						 * the element is meaningless, because it will evict
						 * the one that is just inserted. Only one will survive,
						 * we just give up the failed one */
						goto finish;
					}
				} else {
					/* exceed the maximum insert time, evict one */
					if (b->sig[chosen_simd] == sig) {
						if (simd_lane == chosen_simd) {
							b->loc[simd_lane] = loc;
                        }
					}
					/* whether or not succesfully inserted, finish */
					goto finish;
				}
			}
		}

finish:
		;
		//now we get to the next element
	}

	return;
}

__global__ void hash_delete(
		delem_t			*in,
		bucket_t		*hash_table,
		int				total_elem_num,
		int				thread_num)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x; 
	int id = 0;
	// 16 threads to cooperate for one element
	int step = thread_num >> ELEM_NUM_P;
	int ballot;
	
	int simd_lane = idx & ((1 << ELEM_NUM_P) - 1);
	int elem_id = idx >> ELEM_NUM_P;
	bucket_t *b;

	int bit_move;
	bit_move = idx & (((1 << (5 - ELEM_NUM_P)) - 1) << ELEM_NUM_P);

	for (id = elem_id; id < total_elem_num; id += step) {
		delem_t *elem = &(in[id]);

		b = &(hash_table[elem->hash & HASH_MASK]);
		/* first perform ballot */
		ballot = __ballot_sync(FULL_MASK, b->sig[simd_lane] == elem->sig && b->loc[simd_lane] == elem->loc);

		if (b->sig[simd_lane] == elem->sig && b->loc[simd_lane] == elem->loc) {
			b->sig[simd_lane] = 0;
		}

		ballot = (ballot >> bit_move) & ((1 << ELEM_NUM) - 1);
		if (ballot != 0) {
			continue;
		}

		//b = &(hash_table[(elem->hash ^ elem->sig) & HASH_MASK]);
		int hash = (((elem->hash ^ elem->sig) & BLOCK_HASH_MASK) 
				| (elem->hash & ~BLOCK_HASH_MASK)) & HASH_MASK; 
		b = &(hash_table[hash]);
		if (b->sig[simd_lane] == elem->sig && b->loc[simd_lane] == elem->loc) {
			b->sig[simd_lane] = 0;
		}
	}

	return;
}

extern "C" void gpu_hash_search(
		selem_t 	*in,
		loc_t		*out,
		bucket_t	*hash_table,
		int			num_elem,
		int 		num_thread,
		int			threads_per_blk,
		hipStream_t stream)
{
	int num_blks = (num_thread + threads_per_blk - 1) / threads_per_blk;
	assert(num_thread > threads_per_blk);
	assert(threads_per_blk <= 1024);
	//assert(num_thread <= num_elem);
	if (num_thread % 32 != 0) {
		num_thread = (num_thread + 31) & 0xffe0;
	}
	assert(num_thread % 32 == 0);

	/* prefer L1 cache rather than shared memory,
	   the other is hipFuncCachePreferShared
	*/
	//void (*funcPtr)(selem_t *, loc_t *, bucket_t *, int, int);
	//funcPtr = hash_search;
	//hipFuncSetCacheConfig(reinterpret_cast<const void*>(*funcPtr), hipFuncCachePreferL1);
	

	//printf("stream=%d, threads_per_blk =%d, num_blks = %d\n", stream, threads_per_blk, num_blks);
	if (stream == 0) {
		hash_search<<<num_blks, threads_per_blk>>>(
			in, out, hash_table, num_elem, num_thread);
	} else  {
		hash_search<<<num_blks, threads_per_blk, 0, stream>>>(
			in, out, hash_table, num_elem, num_thread);
	}

	return;
}

/* num_blks is the array size of blk_input and blk_output */
extern "C" void gpu_hash_insert(
		bucket_t	*hash_table,
		ielem_t		**blk_input,
		int			*blk_elem_num,
		int			num_blks,
		hipStream_t stream)
{
	int threads_per_blk = 32;
	//printf("hash_insert: num_blks %d, threads_per_blk %d\n", num_blks, threads_per_blk);

	// prefer L1 cache rather than shared cache
	//void (*funcPtr)(bucket_t *, ielem_t **, loc_t **, int *);
	//funcPtr = hash_insert;
	//hipFuncSetCacheConfig(reinterpret_cast<const void*>(*funcPtr), hipFuncCachePreferL1);
	assert(ELEM_NUM_P < 5 && ELEM_NUM_P > 0);

#if defined(HASH_2CHOICE)
	if (stream == 0) {
		hash_insert_2choice<<<num_blks, threads_per_blk>>>(
			hash_table, blk_input, blk_elem_num);
	} else {
		hash_insert_2choice<<<num_blks, threads_per_blk, 0, stream>>>(
			hash_table, blk_input, blk_elem_num);
	}
#elif defined(HASH_CUCKOO)
	if (stream == 0) {
        hash_insert_cuckoo<<<num_blks, threads_per_blk>>>(
			hash_table, blk_input, blk_elem_num);
	} else {
		hash_insert_cuckoo<<<num_blks, threads_per_blk, 0, stream>>>(
			hash_table, blk_input, blk_elem_num);
	}
#endif

	return;
}

extern "C" void gpu_hash_delete(
		delem_t 	*in,
		bucket_t	*hash_table,
		int			num_elem,
		int 		num_thread,
		int			threads_per_blk,
		hipStream_t stream)
{
	int num_blks = (num_thread + threads_per_blk - 1) / threads_per_blk;
	assert(num_thread >= threads_per_blk);
	assert(threads_per_blk <= 1024);
	//assert(num_thread <= num_elem);
	if (num_thread % 32 != 0) {
		num_thread = (num_thread + 31) & 0xffe0;
	}
	assert(num_thread % 32 == 0);

	/* prefer L1 cache rather than shared memory,
	   the other is hipFuncCachePreferShared
	*/
	//void (*funcPtr)(selem_t *, loc_t *, bucket_t *, int, int);
	//funcPtr = hash_search;
	//hipFuncSetCacheConfig(reinterpret_cast<const void*>(*funcPtr), hipFuncCachePreferL1);
	

	//printf("stream=%d, threads_per_blk =%d, num_blks = %d\n", stream, threads_per_blk, num_blks);
	if (stream == 0) {
		hash_delete<<<num_blks, threads_per_blk>>>(
			in, hash_table, num_elem, num_thread);
	} else  {
		hash_delete<<<num_blks, threads_per_blk, 0, stream>>>(
			in, hash_table, num_elem, num_thread);
	}

	return;
}
